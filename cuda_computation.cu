#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "sunset.h"
#include <stdio.h>

__global__ void computeValueKernel(const double* lat, const double* lng, const int* guessHour, double* output, int numEntries) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEntries) {
        double temp = 0;
        // Added complexity: Perform multiple operations in a loop
        for (int i = 0; i < 10000; ++i) {
            temp += sin(lat[idx]) * cos(lng[idx]) * tan(0.1 * guessHour[idx]) - log(1 + fabs(lat[idx]));
            temp *= 1.1; // Arbitrary multiplier for added complexity
        }
        output[idx] = temp;
    }
}

extern "C" void runCudaComputation(const sunset* sunsets, double* output, int numEntries) {
    double *d_lat, *d_lng;
    int *d_guessHour;
    double *d_output;

    hipMalloc(&d_lat, numEntries * sizeof(double));
    hipMalloc(&d_lng, numEntries * sizeof(double));
    hipMalloc(&d_guessHour, numEntries * sizeof(int));
    hipMalloc(&d_output, numEntries * sizeof(double));

    // Properly copy data to device arrays
    for (int i = 0; i < numEntries; ++i) {
        hipMemcpy(&d_lat[i], &sunsets[i].lat, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(&d_lng[i], &sunsets[i].lng, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(&d_guessHour[i], &sunsets[i].guessHour, sizeof(int), hipMemcpyHostToDevice);
    }

    int blockSize = 32;
    int numBlocks = (numEntries + blockSize - 1) / blockSize;
    computeValueKernel<<<numBlocks, blockSize>>>(d_lat, d_lng, d_guessHour, d_output, numEntries);

    // Add error checking
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        // Handle error...
    }

    hipMemcpy(output, d_output, numEntries * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_lat);
    hipFree(d_lng);
    hipFree(d_guessHour);
    hipFree(d_output);
}
